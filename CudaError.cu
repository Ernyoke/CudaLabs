#include "CudaError.h"
#include <iostream>

void cudaerror::CheckFatal(hipError_t error) {
	if (error != hipSuccess) {
		std::cerr << hipGetErrorString(error) << " in " << __FILE__ << " at line " << __LINE__ << std::endl;
		exit(EXIT_FAILURE);
	}
}

void cudaerror::CheckNonFatal(hipError_t error) {
	if (error != hipSuccess) {
		std::cout << hipGetErrorString(error) << " in " << __FILE__ << " at line " << __LINE__ << std::endl;
	}
}
