/*
 * Lab1.cpp
 *
 *  Created on: Sep 30, 2016
 *      Author: ervin
 */

#include "Lab1.h"
#include <hip/hip_runtime.h>
#include <iostream>

Lab1::Lab1() {
	// TODO Auto-generated constructor stub

}

Lab1::~Lab1() {
	// TODO Auto-generated destructor stub
}

void Lab1::Dump() {
	hipGetDeviceCount(&deviceCount);
	std::cout << "Number of CUDA devices:" << deviceCount << std:: endl;

	for (unsigned i = 0; i < deviceCount; ++i) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, i);

		std::cout << "Device name:" << deviceProp.name << std::endl;
		std::cout << "Total global memory: " << deviceProp.totalGlobalMem / 1024 / 1024 << " MB" << std::endl;
		std::cout << "Shared memory per block: " << deviceProp.sharedMemPerBlock / 1024 << " KB" << std::endl;
		std::cout << "Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
		std::cout << "Clock Rate: " << deviceProp.clockRate << " KHz" << std::endl;
		std::cout << "Maximum grid size: " << deviceProp.maxGridSize[0] << " " << deviceProp.maxGridSize[1] << " " << deviceProp.maxGridSize[2] << " " << std::endl;

	}
}

