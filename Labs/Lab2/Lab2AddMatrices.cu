#include "hip/hip_runtime.h"
/*
 * Lab2AddMatrices.cpp
 *
 *  Created on: Sep 30, 2016
 *      Author: ervin
 */

#include "Lab2AddMatrices.h"
#include "../../Utilities.h"
#include <hip/hip_runtime.h>

Lab2_AddMatrices::Lab2_AddMatrices(int *a, int *b, unsigned n, unsigned m) :
		h_a(a), h_b(b), n(n), m(m) {
}

Lab2_AddMatrices::~Lab2_AddMatrices() {
	// TODO Auto-generated destructor stub
}

__global__ void add(int *d_a, const int *d_b) {
	d_a[blockDim.x * blockIdx.x + threadIdx.x] += d_b[blockDim.x * blockIdx.x
			+ threadIdx.x];
}

void Lab2_AddMatrices::Dump() {
	int N = n * m;
	int *d_a = NULL;
	int *d_b = NULL;

	hipMalloc((void**) &d_a, sizeof(int) * N);
	hipMalloc((void**) &d_b, sizeof(int) * N);

	hipMemcpy(d_a, h_a, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(int) * N, hipMemcpyHostToDevice);

	add<<<n, m>>>(d_a, d_b);

	hipMemcpy(h_a, d_a, sizeof(int) * N, hipMemcpyDeviceToHost);

	utilities::PrintVector(h_a, N);

	hipFree(d_a);
	hipFree(d_b);
}

