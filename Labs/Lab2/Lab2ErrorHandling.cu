/*
 * Lab2ErrorHandling.cpp
 *
 *  Created on: Sep 30, 2016
 *      Author: ervin
 */

#include "Lab2ErrorHandling.h"
#include "../../CudaError.h"
#include <hip/hip_runtime.h>

Lab2_ErrorHandling::Lab2_ErrorHandling() {
	// TODO Auto-generated constructor stub

}

Lab2_ErrorHandling::~Lab2_ErrorHandling() {
	// TODO Auto-generated destructor stub
}

void Lab2_ErrorHandling::Dump() {
	int *d_a = NULL;
	size_t size = 1024 * 1024 * 1024 * 10;
	hipError_t err = hipMalloc((void**)&d_a, size);
	cudaerror::CheckFatal(err);
}

