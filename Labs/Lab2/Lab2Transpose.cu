#include "hip/hip_runtime.h"
/*
 * Lab2Transpose.cpp
 *
 *  Created on: Sep 30, 2016
 *      Author: ervin
 */

#include "Lab2Transpose.h"
#include "../../Utilities.h"
#include <hip/hip_runtime.h>

Lab2_Transpose::Lab2_Transpose(int *a, unsigned n, unsigned m) :
		h_a(a), n(n), m(m) {
}

Lab2_Transpose::~Lab2_Transpose() {
	// TODO Auto-generated destructor stub
}

__global__ void transp(int *d_A, int *d_B) {
	d_A[blockDim.x * threadIdx.x + blockIdx.x] = d_B[blockDim.x * blockIdx.x
			+ threadIdx.x];
}

void Lab2_Transpose::Dump() {
	unsigned N = n * m;
	int *d_a = NULL;
	int *d_b = NULL;

	hipMalloc((void**) &d_a, sizeof(int) * N);
	hipMalloc((void**) &d_b, sizeof(int) * N);

	hipMemcpy(d_b, h_a, sizeof(int) * N, hipMemcpyHostToDevice);

	transp<<<3,3>>>(d_a, d_b);

	hipMemcpy(h_a, d_a, sizeof(int) * N, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);

	utilities::PrintVector(h_a, N);
}
